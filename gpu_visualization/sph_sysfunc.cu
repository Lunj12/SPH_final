#include "hip/hip_runtime.h"
#include "sph_header.h"
#include "sph_system.h"
#include "sph_math.h"
#include "cutil_math.h"

void alloc_array(void **dev_ptr, size_t size)
{
    hipMalloc(dev_ptr, size);
}

void free_array(void *dev_ptr)
{
    hipFree(dev_ptr);
}

void copy_array(void *ptr_a, void *ptr_b, size_t size, int type)
{
	if(type == 1)
	{
		hipMemcpy(ptr_a, ptr_b, size, hipMemcpyHostToDevice);
		return;
	}

	if(type == 2)
	{
		hipMemcpy(ptr_a, ptr_b, size, hipMemcpyDeviceToHost);
		return;
	}

	if(type == 3)
	{
		hipMemcpy(ptr_a, ptr_b, size, hipMemcpyDeviceToDevice);
		return;
	}

	return;
}

void compute_grid_size(uint num_particle, uint block_size, uint &num_blocks, uint &num_threads)
{
    num_threads=min(block_size, num_particle);
    num_blocks=iDivUp(num_particle, num_threads);
}

__global__
void copy_buffer_kernel(Particle *dMem, float2 *buffer, uint num_particle)
{
	uint index=blockIdx.x*blockDim.x+threadIdx.x;

    if(index >= num_particle)
	{
		return;
	}

	buffer[index].x=dMem[index].pos.x;
	buffer[index].y=dMem[index].pos.y;
}

void copy_buffer(Particle *dMem, float2 *buffer, uint num_particle)
{
	if(num_particle == 0)
	{
		return;
	}

	uint num_threads;
	uint num_blocks;

    compute_grid_size(num_particle, 256, num_blocks, num_threads);

	copy_buffer_kernel<<< num_blocks, num_threads >>>(dMem, buffer, num_particle);
}

__device__
int2 calc_grid_pos(float2 p, SysParam* dParam)
{
	int2 grid_pos;
    grid_pos.x = floor((p.x) / dParam->cell_size);
    grid_pos.y = floor((p.y) / dParam->cell_size);

    return grid_pos;
}

__device__ uint calc_grid_hash(int2 grid_pos, SysParam* dParam)
{
	if(grid_pos.x<0 || grid_pos.x>=dParam->grid_size.x || grid_pos.y<0 || grid_pos.y>=dParam->grid_size.y)
	{
		return (uint)0xffffffff;
	}

	grid_pos.x = grid_pos.x & (dParam->grid_size.x-1);
    grid_pos.y = grid_pos.y & (dParam->grid_size.y-1);

	return ((uint)(grid_pos.y))*dParam->grid_size.x+(uint)(grid_pos.x);
}

__global__
void calc_hashK(uint *dHash, uint *dIndex, Particle *dMem, uint num_particle, SysParam* dParam)
{
    uint index=blockIdx.x*blockDim.x+threadIdx.x;

    if(index >= num_particle)
	{
		return;
	}

	int2 grid_pos=calc_grid_pos(dMem[index].pos, dParam);
    uint hash=calc_grid_hash(grid_pos, dParam);

    dHash[index]=hash;
    dIndex[index]=index;
}

void calc_hash(uint *dHash, uint *dIndex, Particle *dMem, uint num_particle, SysParam* dParam)
{
	if(num_particle == 0)
	{
		return;
	}

    uint num_threads;
	  uint num_blocks;

    compute_grid_size(num_particle, 256, num_blocks, num_threads);

	calc_hashK<<< num_blocks, num_threads >>>(dHash, dIndex, dMem, num_particle, dParam);
}

void sort_particles(uint *dHash, uint *dIndex, uint num_particle)
{
	if(num_particle == 0)
	{
		return;
	}

    thrust::sort_by_key(thrust::device_ptr<uint>(dHash),
                        thrust::device_ptr<uint>(dHash + num_particle),
                        thrust::device_ptr<uint>(dIndex));
                        //key_begin,key_end,values
}

__global__
void find_start_end_kernel(uint *dStart, uint *dEnd, uint *dHash, uint *dIndex, uint num_particle)
{
	extern __shared__ uint shared_hash[];
    uint index=blockIdx.x*blockDim.x+threadIdx.x;

    uint hash;

    if(index < num_particle)
	{
        hash=dHash[index];
	    shared_hash[threadIdx.x+1]=hash;

	    if(index > 0 && threadIdx.x == 0)
	    {
		    shared_hash[0]=dHash[index-1];
	    }
	}

	__syncthreads();

	if(index < num_particle)
	{
		if(index == 0 || hash != shared_hash[threadIdx.x])
	    {
		    dStart[hash]=index;

            if(index > 0)
			         {
                dEnd[shared_hash[threadIdx.x]]=index;
			         }
	    }

        if (index == num_particle-1)
        {
            dEnd[hash]=index+1;
        }
	}
}

void find_start_end(uint *dStart, uint *dEnd, uint *dHash, uint *dIndex, uint num_particle, uint num_cell)
{
	if(num_particle == 0)
	{
		return;
	}

    uint num_thread;
	  uint num_block;
    compute_grid_size(num_particle, 256, num_block, num_thread);

    hipMemset(dStart, 0xffffffff, num_cell*sizeof(int));
	hipMemset(dEnd, 0x0, num_cell*sizeof(int));

    uint smemSize=sizeof(int)*(num_thread+1);

    find_start_end_kernel<<< num_block, num_thread, smemSize>>>(dStart, dEnd, dHash, dIndex, num_particle);
}

__global__
void integrate_velocity_kernel(Particle* dMem, uint num_particle_fluid, SysParam* dParam)
{
	uint index=blockIdx.x*blockDim.x+threadIdx.x;

	if(index >= num_particle_fluid)
	{
		return;
	}

	Particle *p=&(dMem[index]);

	p->vel=p->vel+p->acc*dParam->time_step/p->dens+dParam->gravity*dParam->time_step;
	p->pos=p->pos+p->vel*dParam->time_step;

	if(p->pos.x >= dParam->world_size.x-BOUNDARY)
	{
		p->vel.x=p->vel.x*dParam->wall_damping;
		p->pos.x=dParam->world_size.x-BOUNDARY;
	}

	if(p->pos.x < 0.0f)
	{
		p->vel.x=p->vel.x*dParam->wall_damping;
		p->pos.x=0.0f;
	}

	if(p->pos.y >= dParam->world_size.y-BOUNDARY)
	{
		p->vel.y=p->vel.y*dParam->wall_damping;
		p->pos.y=dParam->world_size.y-BOUNDARY;
	}

	if(p->pos.y < 0.0f)
	{
		p->vel.y=p->vel.y*dParam->wall_damping;
		p->pos.y=0.0f;
	}

	p->ev=(p->ev+p->vel)/2;
}

void integrate_velocity(Particle *dMem, uint num_particle_fluid, SysParam* dParam)
{
	if(num_particle_fluid == 0)
	{
		return;
	}

	uint num_thread;
	uint num_block;
    compute_grid_size(num_particle_fluid, 256, num_block, num_thread);

	integrate_velocity_kernel<<< num_block, num_thread >>>(dMem, num_particle_fluid, dParam);
}

__device__
float compute_cell_density(uint index,
					int2 neighbor,
					Particle *dMem,
					uint *dHash,
					uint *dIndex,
					uint *dStart,
					uint *dEnd,
					uint num_particle,
					uint tot_cell,
          SysParam* dParam)
{
	float total_cell_density=0.0f;
	uint grid_hash=calc_grid_hash(neighbor, dParam);
	if(grid_hash == 0xffffffff)
	{
		return total_cell_density;
	}
	uint start_index=dStart[grid_hash];

	float mass=dParam->mass;
	float kernel_2=dParam->kernel_2;
	float poly6_value=dParam->poly6_value;

	float2 rel_pos;
	float r2;

	Particle *p=&(dMem[index]);
	Particle *np;
	uint neighbor_index;

	if(start_index != 0xffffffff)
	{
        uint end_index=dEnd[grid_hash];

        for(uint count_index=start_index; count_index<end_index; count_index++)
		{
			neighbor_index=dIndex[count_index];
			np=&(dMem[neighbor_index]);

			rel_pos=np->pos-p->pos;
			r2=rel_pos.x*rel_pos.x+rel_pos.y*rel_pos.y;

			if(r2<INF || r2>=kernel_2)
			{
				continue;
			}

			total_cell_density=total_cell_density + mass * poly6_value * pow(kernel_2-r2, 3);
        }
	}

	return total_cell_density;
}

__global__
void compute_density_kernel(Particle *dMem,
							uint *dHash,
							uint *dIndex,
							uint *dStart,
							uint *dEnd,
							uint num_particle,
							uint tot_cell,
              SysParam* dParam)
{
	uint index=blockIdx.x*blockDim.x+threadIdx.x;

	if(index >= num_particle)
	{
		return;
	}

	int2 cell_pos=calc_grid_pos(dMem[index].pos, dParam);

	float total_density=0;

	for(int y=-1; y<=1; y++)
	{
		for(int x=-1; x<=1; x++)
		{
			int2 neighbor_pos = cell_pos+ make_int2(x, y);
			total_density=total_density+compute_cell_density(index, neighbor_pos, dMem, dHash, dIndex, dStart, dEnd, num_particle, tot_cell, dParam);
        }
    }
	total_density=total_density+dParam->self_dens;

	if(total_density < INF)
	{
		dMem[index].dens=dParam->rest_density;
	}

	dMem[index].dens=total_density;
	dMem[index].pres=(pow(dMem[index].dens / dParam->rest_density, 7) - 1) * dParam->gas_constant;
}

__device__
float2 compute_cell_force(uint index, int2 neighbor, Particle *dMem, uint *dHash, uint *dIndex, uint *dStart, uint *dEnd, uint num_particle, uint tot_cell, float2 &grad_color, float &lplc_color, SysParam* dParam)
{
	float2 total_cell_force=make_float2(0.0f);
	uint grid_hash=calc_grid_hash(neighbor, dParam);

	if(grid_hash == 0xffffffff)
	{
		return total_cell_force;
	}

	uint start_index=dStart[grid_hash];

	float kernel=dParam->kernel;
	float mass=dParam->mass;
	float kernel_2=dParam->kernel_2;

	uint neighbor_index;

	Particle *p=&(dMem[index]);
	Particle *np;

	float2 rel_pos;
	float r2;
	float r;

	float V;
	float kernel_r;

	float pressure_kernel;
	float temp_force;

	float2 rel_vel;
	float viscosity_kernel;


	if(start_index != 0xffffffff)
	{
		uint end_index=dEnd[grid_hash];

        for(uint count_index=start_index; count_index<end_index; count_index++)
		{
			neighbor_index=dIndex[count_index];

			np=&(dMem[neighbor_index]);

			rel_pos=p->pos-np->pos;
			r2=rel_pos.x*rel_pos.x+rel_pos.y*rel_pos.y;

			if(r2 < kernel_2 && r2 > INF)
			{
				r=sqrt(r2);
				V=mass/np->dens/2;
				kernel_r=kernel-r;

				pressure_kernel=dParam->spiky_value * kernel_r * kernel_r;
				temp_force=V * (p->pres+np->pres) * pressure_kernel;
				total_cell_force=total_cell_force-rel_pos*temp_force/r;

				rel_vel=np->ev-p->ev;
				viscosity_kernel=dParam->visco_value*(kernel-r);
				temp_force=V * dParam->viscosity * viscosity_kernel;
				total_cell_force=total_cell_force + rel_vel*temp_force;

				float temp=(-1) * dParam->grad_poly6 * V * pow(kernel_2-r2, 2);
				grad_color.x += temp * rel_pos.x;
				grad_color.y += temp * rel_pos.y;
				lplc_color += dParam->lplc_poly6 * V * (kernel_2-r2) * (r2-3/4*(kernel_2-r2));
			}
        }
	}

	return total_cell_force;
}

__global__
void compute_force_kernel(Particle *dMem, uint *dHash, uint *dIndex, uint *dStart, uint *dEnd, uint num_particle, uint tot_cell, SysParam* dParam)
{
	uint index=blockIdx.x*blockDim.x+threadIdx.x;

	if(index >= num_particle)
	{
		return;
	}

	int2 cell_pos=calc_grid_pos(dMem[index].pos, dParam);

	float2 total_force=make_float2(0.0f, 0.0f);
	float2 grad_color=make_float2(0.0f);
	float lplc_color=0.0f;

	for(int y=-1; y<=1; y++)
	{
		for(int x=-1; x<=1; x++)
		{
			int2 neighbor_pos = cell_pos + make_int2(x, y);
			total_force=total_force+compute_cell_force(index, neighbor_pos, dMem, dHash, dIndex, dStart, dEnd, num_particle, tot_cell, grad_color, lplc_color, dParam);
		}
	}
	dMem[index].acc=total_force;

	lplc_color+=dParam->self_lplc_color/dMem[index].dens;
	dMem[index].surf_norm=sqrt(grad_color.x*grad_color.x+grad_color.y*grad_color.y);
	float2 force;

	if(dMem[index].surf_norm > dParam->surf_normal)
	{
		force=dParam->surf_coe * lplc_color * grad_color / dMem[index].surf_norm;
	}
	else
	{
		force=make_float2(0.0f, 0.0f);
	}

	dMem[index].acc+=force;
}

void compute(Particle *dMem, uint *dHash, uint *dIndex, uint *dStart, uint *dEnd, uint num_particle, uint tot_cell, SysParam* dParam)
{
	if(num_particle == 0)
	{
		return;
	}

	uint num_thread;
	uint num_block;
    compute_grid_size(num_particle, 256, num_block, num_thread);

	compute_density_kernel<<< num_block, num_thread >>>(dMem, dHash, dIndex, dStart, dEnd, num_particle, tot_cell, dParam);
	compute_force_kernel<<< num_block, num_thread >>>(dMem, dHash, dIndex, dStart, dEnd, num_particle, tot_cell, dParam);
}

__global__
void move_body_kernel(Particle *dMem, uint num_particle, uint num_particle_bc, SysParam* dParam)
{
  uint index=blockIdx.x*blockDim.x+threadIdx.x;
  uint num_particle_body=num_particle-num_particle_bc;

	if(index >= num_particle_body || dParam->center.y<0.2f)
	{
		return;
	}


  	Particle *p=&(dMem[index+num_particle_bc+1]);
    dParam->center.x=0.0f;
	  dParam->center.y+=-0.003f;
    p->pos.x=dParam->center.x+p->pos.x;
   	p->pos.y=-0.003f+p->pos.y;
    p->vel=make_float2(0.0f);
    p->acc=make_float2(0.0f);
    p->ev=make_float2(0.0f);
}


void move_body(Particle *dMem, uint num_particle, uint num_particle_bc, SysParam* dParam)
{
  if(num_particle==num_particle_bc){
    return;
  }

  uint num_thread;
  uint num_block;
  uint num_particle_body=num_particle-num_particle_bc;
    compute_grid_size(num_particle_body, 256, num_block, num_thread);
    move_body_kernel<<< num_block, num_thread >>>(dMem, num_particle, num_particle_bc, dParam);
}

__global__
void fix_boundary_kernel(Particle *dMem, uint num_particle, uint num_particle_fluid, SysParam* dParam)
{
  uint index=blockIdx.x*blockDim.x+threadIdx.x;

  if(index >= num_particle_fluid)
  {
    return;
  }


    Particle *p=&(dMem[index+num_particle_fluid+1]);
    p->vel=make_float2(0.0f);
    p->vel=make_float2(0.0f);
    p->acc=make_float2(0.0f);
    p->ev=make_float2(0.0f);
}

void fix_boundary(Particle *dMem, uint num_particle, uint num_particle_fluid, SysParam* dParam)
{
  if(num_particle==num_particle_fluid){
    return;
  }

  uint num_thread;
  uint num_block;
  uint num_particle_bc=num_particle-num_particle_fluid;
    compute_grid_size(num_particle_bc, 256, num_block, num_thread);
    fix_boundary_kernel<<< num_block, num_thread >>>(dMem, num_particle, num_particle_fluid, dParam);

}
